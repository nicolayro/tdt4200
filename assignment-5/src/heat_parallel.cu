#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <sys/time.h>

#include "../inc/argument_utils.h"

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

typedef int64_t int_t;
typedef double real_t;

int_t
    M,
    N,
    max_iteration,
    snapshot_frequency;

real_t
    *h_temp[2] = { NULL, NULL },
    *h_thermal_diffusivity,
    // TODO 1: Declare device side pointers to store host-side data.
    *d_temp = NULL,
    *d_temp_next = NULL,
    *d_thermal_diffusivity,
    dt;


#define T(x,y)                      d_temp[(y) * (N + 2) + (x)]
#define T_next(x,y)                 d_temp_next[(y) * (N + 2) + (x)]
#define THERMAL_DIFFUSIVITY(x,y)    d_thermal_diffusivity[(y) * (N + 2) + (x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void time_step (real_t *d_temp, real_t *d_temp_next, real_t *d_thermal_diffusivity, real_t dt, int_t N, int_t M);
__device__ void boundary_condition(real_t *d_temp, real_t *d_temp_next, int_t N, int_t M);
void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );


void
swap ( real_t** m1, real_t** m2 )
{
    real_t* tmp;
    tmp = *m1;
    *m1 = *m2;
    *m2 = tmp;
}


int
main ( int argc, char **argv )
{
    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    M = options->M;
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(N / threadsPerBlock.x, M / threadsPerBlock.y);

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        // TODO 6: Launch the time_step-kernel.
        time_step<<<numBlocks, threadsPerBlock>>>(d_temp, d_temp_next, d_thermal_diffusivity, dt, N, M);

        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // TODO 8: Copy data from device to host.
            domain_save ( iteration );
        }

        // swap( &h_temp[0], &h_temp[1] );
        // TODO 7: Swap device pointers.
        swap( &d_temp, &d_temp_next );
    }

    gettimeofday ( &t_end, NULL );
    printf ( "Total elapsed time: %lf seconds\n",
            WALLTIME(t_end) - WALLTIME(t_start)
            );


    domain_finalize();

    exit ( EXIT_SUCCESS );
}


// TODO 4: Make time_step() a CUDA kernel
//         where one thread is responsible for one grid point.
__global__
void
time_step (real_t *d_temp, real_t *d_temp_next, real_t *d_thermal_diffusivity, real_t dt, int_t N, int_t M)
{
    real_t c, t, b, l, r, K, new_value;

    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (x > N || y > M) {
        return;
    }

    boundary_condition(d_temp, d_temp_next, N, M);

    c = T(x, y);

    t = T(x - 1, y);
    b = T(x + 1, y);
    l = T(x, y - 1);
    r = T(x, y + 1);
    K = THERMAL_DIFFUSIVITY(x, y);

    new_value = c + K * dt * ((l - 2 * c + r) + (b - 2 * c + t));

    T_next(x, y) = new_value;

}

// TODO 5: Make boundary_condition() a device function and
//         call it from the time_step-kernel.
//         Chose appropriate threads to set the boundary values.
__device__
void
boundary_condition(real_t *d_temp, real_t *d_temp_next, int_t N, int_t M)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (y == 1) {
        T(x, 0) = T(x, 2);
    }

    if (y == M) {
        T(x, M+1) = T(x, M-1);
    }

    if (x == 1) {
        T(0, y) = T(2, y);
    }

    if (x == N) {
        T(N+1, y) = T(N-1, y);
    }
    
}


void
domain_init ( void )
{
    size_t size = (M+2) * (N+2) * sizeof(real_t);

    h_temp[0] = (real_t*) malloc(size);
    h_temp[1] = (real_t*) malloc(size);
    h_thermal_diffusivity = (real_t*) malloc(size);

    // TODO 2: Allocate device memory.
    hipMalloc(&d_temp, size);
    hipMalloc(&d_temp_next, size);
    hipMalloc(&d_thermal_diffusivity, size);

    dt = 0.1;

    for ( int_t y = 1; y <= M; y++ )
    {
        for ( int_t x = 1; x <= N; x++ )
        {
            real_t temperature = 30 + 30 * sin((x + y) / 20.0);
            real_t diffusivity = 0.05 + (30 + 30 * sin((N - x + y) / 20.0)) / 605.0;

            h_temp[0][ y*(N+2) + x ] = temperature;
            h_temp[1][ y*(N+2) + x ] = temperature;
            h_thermal_diffusivity[ y*(N+2) + x ] = diffusivity;
        }
    }

    // TODO 3: Copy data from host to device.
    hipMemcpy(d_temp, h_temp[0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_temp_next, h_temp[1], size, hipMemcpyHostToDevice);
    hipMemcpy(d_thermal_diffusivity, h_thermal_diffusivity, size, hipMemcpyHostToDevice);
}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    size_t size = (M+2) * (N+2) * sizeof(real_t);
    hipMemcpy(h_temp[0], d_temp, size, hipMemcpyDeviceToHost);

    FILE *out = fopen ( filename, "wb" );
    if ( ! out ) {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    for ( int_t iter = 1; iter <= N; iter++)
    {
        fwrite( h_temp[0] + (M+2) * iter + 1, sizeof(real_t), N, out );
    }
    fclose ( out );
}


void
domain_finalize ( void )
{
    free ( h_temp[0] );
    free ( h_temp[1] );
    free ( h_thermal_diffusivity );

    // TODO 9: Free device memory.
    hipFree(d_temp);
    hipFree(d_temp_next);
    hipFree(d_thermal_diffusivity);
}
